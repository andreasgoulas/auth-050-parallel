#include "hip/hip_runtime.h"
#include "ising.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define TILE_SIZE 32  // Points per block
#define BLOCK_SIZE 8  // Threads per block

#define CHECK(x) do {                                \
    hipError_t err = (x);                           \
    if (err != hipSuccess) {                        \
      printf("error %s\n", hipGetErrorString(err)); \
    }                                                \
  } while(0);

__global__
void compute(int* g_prev, int* g_next, double* w, int n) {
  int x0 = TILE_SIZE * blockIdx.x;
  int y0 = TILE_SIZE * blockIdx.y;
  for (int y = y0 + threadIdx.y; y < y0 + TILE_SIZE; y += BLOCK_SIZE) {
    for (int x = x0 + threadIdx.x; x < x0 + TILE_SIZE; x += BLOCK_SIZE) {
      if (x >= n || y >= n) {
        continue;
      }

      double sum = 0.0;
      for (int dy = 0; dy < 5; ++dy) {
        for (int dx = 0; dx < 5; ++dx) {
          int xx = (x + dx - 2 + n) % n;
          int yy = (y + dy - 2 + n) % n;
          sum += w[dx + 5 * dy] * g_prev[xx + n * yy];
        }
      }

      int v;
      if (sum > 1e-6) {
        v = 1;
      } else if (sum < -1e-6) {
        v = -1;
      } else {
        v = g_prev[x + n * y];
      }

      g_next[x + n * y] = v;
    }
  }
}

extern "C"
void ising(int* g, double* w, int k, int n) {
  double* dev_w;
  int* dev_g_prev, *dev_g_next;
  CHECK(hipMalloc(&dev_w, 5 * 5 * sizeof(double)));
  CHECK(hipMalloc(&dev_g_prev,n * n * sizeof(int)));
  CHECK(hipMalloc(&dev_g_next, n * n * sizeof(int)));

  CHECK(hipMemcpy(dev_w, w, 5 * 5 * sizeof(double), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(dev_g_prev, g, n * n * sizeof(int), hipMemcpyHostToDevice));

  int num_blocks = (n + TILE_SIZE - 1) / TILE_SIZE;
  uint3 dim_block = make_uint3(BLOCK_SIZE, BLOCK_SIZE, 1);
  uint3 dim_grid = make_uint3(num_blocks, num_blocks, 1);

  for (int i = 0; i < k; ++i) {
    compute<<<dim_grid, dim_block>>>(dev_g_prev, dev_g_next, dev_w, n);
    CHECK(hipGetLastError());

    int* tmp = dev_g_prev;
    dev_g_prev = dev_g_next;
    dev_g_next = tmp;
  }

  CHECK(hipMemcpy(g, dev_g_prev, n * n * sizeof(int), hipMemcpyDeviceToHost));

  hipFree(dev_w);
  hipFree(dev_g_prev);
  hipFree(dev_g_next);
}
