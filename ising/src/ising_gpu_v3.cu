#include "hip/hip_runtime.h"
#include "ising.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define TILE_SIZE 96   // Points per block
#define BLOCK_SIZE 16  // Threads per block

#define CHECK(x) do {                                \
    hipError_t err = (x);                           \
    if (err != hipSuccess) {                        \
      printf("error %s\n", hipGetErrorString(err)); \
    }                                                \
  } while(0);

__global__
void compute(int* g_prev, int* g_next, double* w, int n) {
  int x0 = TILE_SIZE * blockIdx.x;
  int y0 = TILE_SIZE * blockIdx.y;

  __shared__ double w_cache[5][5];
  if (threadIdx.x < 5 && threadIdx.y < 5) {
    w_cache[threadIdx.x][threadIdx.y] = w[threadIdx.x + 5 * threadIdx.y];
  }

  __shared__ int g_cache[TILE_SIZE + 4][TILE_SIZE + 4];
  for (int y = threadIdx.y; y < TILE_SIZE + 4; y += BLOCK_SIZE) {
    for (int x = threadIdx.x; x < TILE_SIZE + 4; x += BLOCK_SIZE) {
      int xx = (x0 + x - 2 + n) % n;
      int yy = (y0 + y - 2 + n) % n;
      g_cache[x][y] = g_prev[xx + n * yy];
    }
  }

  __syncthreads();

  for (int y = y0 + threadIdx.y; y < y0 + TILE_SIZE; y += BLOCK_SIZE) {
    for (int x = x0 + threadIdx.x; x < x0 + TILE_SIZE; x += BLOCK_SIZE) {
      if (x >= n || y >= n) {
        continue;
      }

      int cache_x = (x % TILE_SIZE) + 2;
      int cache_y = (y % TILE_SIZE) + 2;

      double sum = 0.0;
      for (int dy = 0; dy < 5; ++dy) {
        for (int dx = 0; dx < 5; ++dx) {
          int xx = cache_x + dx - 2;
          int yy = cache_y + dy - 2;
          sum += w_cache[dx][dy] * g_cache[xx][yy];
        }
      }

      int v;
      if (sum > 1e-6) {
        v = 1;
      } else if (sum < -1e-6) {
        v = -1;
      } else {
        v = g_cache[cache_x][cache_y];
      }

      g_next[x + n * y] = v;
    }
  }
}

extern "C"
void ising(int* g, double* w, int k, int n) {
  double* dev_w;
  int* dev_g_prev, *dev_g_next;
  CHECK(hipMalloc(&dev_w, 5 * 5 * sizeof(double)));
  CHECK(hipMalloc(&dev_g_prev,n * n * sizeof(int)));
  CHECK(hipMalloc(&dev_g_next, n * n * sizeof(int)));

  CHECK(hipMemcpy(dev_w, w, 5 * 5 * sizeof(double), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(dev_g_prev, g, n * n * sizeof(int), hipMemcpyHostToDevice));

  int num_blocks = (n + TILE_SIZE - 1) / TILE_SIZE;
  uint3 dim_block = make_uint3(BLOCK_SIZE, BLOCK_SIZE, 1);
  uint3 dim_grid = make_uint3(num_blocks, num_blocks, 1);

  for (int i = 0; i < k; ++i) {
    compute<<<dim_grid, dim_block>>>(dev_g_prev, dev_g_next, dev_w, n);
    CHECK(hipGetLastError());

    int* tmp = dev_g_prev;
    dev_g_prev = dev_g_next;
    dev_g_next = tmp;
  }

  CHECK(hipMemcpy(g, dev_g_prev, n * n * sizeof(int), hipMemcpyDeviceToHost));

  hipFree(dev_w);
  hipFree(dev_g_prev);
  hipFree(dev_g_next);
}
